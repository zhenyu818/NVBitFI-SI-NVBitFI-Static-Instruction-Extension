#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdint.h>
#include <stdio.h>

#include "utils/utils.h"
#include "arch.h"

// Global counters are incremented once per warp 
extern "C" __device__ __noinline__ void count_instrs(uint64_t pcounters, int index, int grp_index, int predicate, int num_counters,uint64_t pstatic_inst_threads_count, int static_inst_id, uint64_t pstatic_inst_type) {    
	uint64_t *counters = (uint64_t*)pcounters;
	uint64_t *static_inst_threads_count = (uint64_t *)pstatic_inst_threads_count;
    uint64_t *static_inst_type = (uint64_t *)pstatic_inst_type;

	// Optimization: Instead of all the threads in a warp performing atomicAdd,
	// let's count the number of active threads with predicate=1 in a warp and let just one thread
	// (leader) in the warp perform the atomicAdd
	const int active_mask = __ballot_sync(__activemask(), 1);
	const int leader = __ffs(active_mask) - 1;
	const int laneid = get_laneid();

	// compute the predicate mask
	const int predicate_mask = __ballot_sync(__activemask(), predicate);
	const int num_threads = __popc(predicate_mask);

	if (laneid == leader) { // Am I the leader thread
		atomicAdd((unsigned long long *)&static_inst_threads_count[static_inst_id], num_threads);
        atomicExch((unsigned long long *)&static_inst_type[static_inst_id], grp_index);
		atomicAdd((unsigned long long *)&counters[index], num_threads);
		atomicAdd((unsigned long long *)&counters[NUM_ISA_INSTRUCTIONS+grp_index], num_threads);
		atomicAdd((unsigned long long *)&counters[num_counters-2], num_threads*(grp_index != G_NODEST));
		atomicAdd((unsigned long long *)&counters[num_counters-1], num_threads*(1 - ((grp_index == G_NODEST) || (grp_index == G_PR))));
	}
}
